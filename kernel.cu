/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float* C)
{
    /********************************************************************
     *
     * Compute C = A x B
     *   where A is a (m x k) matrix
     *   where B is a (k x n) matrix
     *   where C is a (m x n) matrix
     *
     ********************************************************************/

    // INSERT KERNEL CODE HERE
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int size = m * n;

    if(row < m && col < n)
    {
        int sum = 0;

        for(int i = 0;i < size;i++)
        {
            sum += A[row * k + i] * B[i * n + col];
        }

        C[row * n + col] = sum;
    }
}

void basicSgemm(char transa, char transb, int m, int n, int k, float alpha, const float *A, int lda, const float *B, int ldb, float beta, float *C, int ldc)
{
    if ((transa != 'N') && (transa != 'n'))
    {
        printf("unsupported value of 'transa'\n");
        return;
    }

    if ((transb != 'N') && (transb != 'n'))
    {
        printf("unsupported value of 'transb'\n");
        return;
    }

    if ((alpha - 1.0f > 1e-10) || (alpha - 1.0f < -1e-10))
    {
        printf("unsupported value of alpha\n");
        return;
    }

    if ((beta - 0.0f > 1e-10) || (beta - 0.0f < -1e-10))
    {
        printf("unsupported value of beta\n");
        return;
    }

    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = 16; // Use 16x16 thread blocks

    //INSERT CODE HERE - UNSURE IF CORRECT; REVIEW IF VERIFY ERRORS OCCUR
    int C_size = (m * n);
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    int numberOfBlocks = ceil(C_size / BLOCK_SIZE);
    dim3 gridDim(numberOfBlocks, numberOfBlocks);

    // Invoke CUDA kernel -----------------------------------------------------

    //INSERT CODE HERE
    mysgemm<<<gridDim,blockDim>>>(m, n, k, A, B, C);

}
